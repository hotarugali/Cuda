
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu(void) {
    printf("Hello world from GPU!\n");
}

int main(void) {
    // hello world from cpu
    printf("Hello world from CPU!\n");
    // hello world from gpu
    hello_from_gpu<<<1,10>>>();
    hipDeviceReset();
    return 0;
}